#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "SequenceAlignment.hpp"


enum DIR { LEFT, DIAG, TOP};


__global__ void alignSequenceGlobalCUDA(const char *textBytes, const uint64_t textNumBytes,
                                        const char *patternBytes, const uint64_t patternNumBytes,
                                        const short *scoreMatrix, const int alphabetSize,
                                        const short gapPenalty, const int numRows, const int numCols,
                                        char *M, int *finalScore)
{
    extern __shared__ int _shared[];
    int *_thisScores = _shared;
    int *_prevScores = _shared + numCols;

    const int tid = threadIdx.x;

    // Each thread copies one text letter.
    const char textByte = (tid > 0) ? textBytes[tid - 1] : alphabetSize;
    // Init first row.
    _thisScores[tid] = tid * -gapPenalty;
    M[tid] = DIR::LEFT;

    __syncthreads();

    // Dynamic programming loop.
    auto thisRowM = M + numCols;
    for (int i_pattern = 1; i_pattern < numRows; ++i_pattern)
    {
        // Advance one row.
         int *tmp = _thisScores;
        _thisScores = _prevScores;
        _prevScores = tmp;

        if (tid == 0)
        {
            _thisScores[tid] = -(i_pattern * gapPenalty);
            thisRowM[0] = DIR::TOP;
            continue;
        }

        const char patternByte = patternBytes[i_pattern - 1];
        const int scoreMatrixIdx = ((int) textByte) * alphabetSize + ((int) patternByte);

        // We are accessing the previous row - wait for all columns to finish.
        const int fromTopScore = _prevScores[tid] - gapPenalty;
        const int fromDiagScore = _prevScores[tid - 1] + scoreMatrix[scoreMatrixIdx];

        const bool isDiagGreaterThanTop = (fromDiagScore > fromTopScore);
        const int maxFromPrev = isDiagGreaterThanTop ? fromDiagScore : fromTopScore;
        const auto tmpDir = isDiagGreaterThanTop ? DIR::DIAG : DIR::TOP;

        for (int i_text = 1; i_text < numCols; ++i_text)
        {
            // We are accessing the previous column within a row.
            if (tid == i_text)
            {
                const int fromLeftScore = _thisScores[tid - 1] - gapPenalty;
                const bool isPrevGreater = (maxFromPrev > fromLeftScore);

                _thisScores[tid] = isPrevGreater ? maxFromPrev : fromLeftScore;
                thisRowM[i_text] = isPrevGreater ? tmpDir : DIR::LEFT;
            }
            __syncthreads();
        }

        thisRowM += numCols;
    }

    if (tid == (numCols - 1))
        finalScore[0] = _thisScores[tid];
}


void SequenceAlignment::alignSequenceGlobalGPU(const SequenceAlignment::Request &request,
                                               SequenceAlignment::Response *response)
{
    const unsigned int numCols = request.textNumBytes + 1;
    const unsigned int numRows = request.patternNumBytes + 1;
    char *M;

    /** Allocate host memory */
    try
    {
        M = new char[numRows * numCols];
        response->alignedTextBytes = new char[2 * request.textNumBytes];
        response->alignedPatternBytes = new char[2 * request.textNumBytes];

    }
    catch(const std::bad_alloc& e)
    {
        std::cerr << SequenceAlignment::MEM_ERROR;
        return;
    }
    /** End Allocate host memory */

    int *d_finalScore;
    short *d_scoreMatrix;
    char *d_textBytes, *d_patternBytes, *d_M;

    /** Allocate and transfer memory */
    if (hipMalloc(&d_finalScore, sizeof(int)) != hipSuccess ||
        hipMalloc(&d_scoreMatrix, sizeof(short) * request.alphabetSize * request.alphabetSize) != hipSuccess ||
        hipMalloc(&d_M, numRows * numCols) != hipSuccess ||
        hipMalloc(&d_textBytes, request.textNumBytes) != hipSuccess ||
        hipMalloc(&d_patternBytes, request.patternNumBytes) != hipSuccess)
    {
        std::cout << MEM_ERROR << std::endl;
        hipFree(d_finalScore);
        hipFree(d_scoreMatrix);
        hipFree(d_M);
        hipFree(d_textBytes);
        hipFree(d_patternBytes);
        return;
    }

    if (hipMemcpy(d_textBytes, request.textBytes, request.textNumBytes, hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_patternBytes, request.patternBytes, request.patternNumBytes, hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_scoreMatrix, request.scoreMatrix, sizeof(short) * (request.alphabetSize * request.alphabetSize), hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cout << MEM_ERROR << std::endl;
        hipFree(d_finalScore);
        hipFree(d_scoreMatrix);
        hipFree(d_M);
        hipFree(d_textBytes);
        hipFree(d_patternBytes);
        return;
    }
    /** End Allocate and transfer memory */

    // this and prev row scores
    const unsigned int sharedMemSize = 2 * numCols * sizeof(int);
    // std::cout << "Num col: " << numCols << "\n";
    // std::cout << "Num bytes in shared: " << sharedMemSize << "\n";
    // std::cout << "Num cols: " << numCols << "\n";

    alignSequenceGlobalCUDA<<<1, numCols, sharedMemSize>>>(d_textBytes, request.textNumBytes,
                                                           d_patternBytes, request.patternNumBytes,
                                                           d_scoreMatrix, request.alphabetSize,
                                                           request.gapPenalty, numRows, numCols,
                                                           d_M, d_finalScore);

    if (hipMemcpy(&(response->score), (d_finalScore), sizeof(int), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(M, d_M, numRows*numCols, hipMemcpyDeviceToHost) != hipSuccess)
    {
        std::cout << "Could not copy back to host memory" << std::endl;
        return;
    }

    traceBack(M, numRows, numCols, request, response);

    hipFree(d_finalScore);
    hipFree(d_scoreMatrix);
    hipFree(d_M);
    hipFree(d_textBytes);
    hipFree(d_patternBytes);
    delete [] M;

    // std::cout << "# Score: \t" << response->score << "\n";

}
