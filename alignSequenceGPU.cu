#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "SequenceAlignment.hpp"

constexpr unsigned int MAX_THREADS_PER_BLOCK = 1024;


enum COLUMN_STATE { UNDEFINED, READY_FOR_0, READY_FOR_1, };

__device__ void yield(char* columnState, const int worker_id, const int col)
{
    columnState[col] = (worker_id == 0) ? COLUMN_STATE::READY_FOR_1 : COLUMN_STATE::READY_FOR_0;
}
__device__ void busy_wait(const char* columnState, const int worker_id, const int col)
{
    auto waitingFor = (worker_id == 0) ? COLUMN_STATE::READY_FOR_0 : COLUMN_STATE::READY_FOR_1;

    while (columnState[col] != waitingFor) continue;
}


__global__ void cuda_fillMatrixNW(const char *textBytes, const char *patternBytes,
                                  const int *scoreMatrix, const int alphabetSize,
                                  const int gapPenalty, const int startRow, const int endRow,
                                  const int numCols, const int workerId, char *columnState,
                                  char *M, int *finalScore)
{
    using SequenceAlignment::DIR;

    const int numRows = endRow - startRow;

    extern __shared__ int _shared[];
    int *_scoreMatrix = _shared;
    int *_thisScores = _shared + alphabetSize*alphabetSize;
    int *_prevScores = _thisScores + numRows;
    int *_prevPrevScores = _prevScores + numRows;

    const int tid = threadIdx.x;

    // Transfer score matrix to shared memory.
    for (int offset=0; offset < alphabetSize*alphabetSize; offset += blockDim.x)
    {
        if ((offset + tid) < alphabetSize*alphabetSize)
            _scoreMatrix[offset + tid] = scoreMatrix[offset + tid];
    }

    const char patternByte = patternBytes[max(0, tid - 1 + startRow)];

    __syncthreads();

    // First half of matrix filling
    int diag_size = 0;
    for (int i_text = 0; i_text < numCols; ++i_text)
    {
        // Advance one diag.
        diag_size = min(diag_size+1, numRows);
        auto tmp = _prevScores;
        _prevScores = _thisScores;
        _thisScores = _prevPrevScores;
        _prevPrevScores = tmp;

        const int threadInRowIdx = i_text - tid;

        if (startRow > 0)
            busy_wait(columnState, workerId, i_text);

        if (tid == 0)
        {
            // First row.
            _thisScores[tid] = -(i_text * gapPenalty);
            M[i_text] = DIR::LEFT;
        }
        else if (tid == (diag_size-1) && i_text < numRows)
        {
            // First column.
            _thisScores[tid] = -(tid * gapPenalty);
            M[tid * numCols] = DIR::TOP;
        }
        else if (tid < diag_size)
        {
            const char textByte = textBytes[threadInRowIdx - 1];
            const int scoreMatrixIdx = ((int) textByte) * alphabetSize + ((int) patternByte);

            const int fromLeftScore = _prevScores[tid] - gapPenalty;
            const int fromTopScore = _prevScores[tid - 1] - gapPenalty;
            const int fromDiagScore = _prevPrevScores[tid - 1] + _scoreMatrix[scoreMatrixIdx];

            const int maxWithGap = max(fromLeftScore, fromTopScore);
            _thisScores[tid] = max(maxWithGap, fromDiagScore);

            const auto dirWithGap = (fromTopScore > fromLeftScore) ? DIR::TOP : DIR::LEFT;
            M[tid*numCols + threadInRowIdx] = (fromDiagScore > maxWithGap) ? DIR::DIAG : dirWithGap;
        }

        if (tid == (diag_size-1) && diag_size == numRows)
            yield(columnState, workerId, threadInRowIdx);

        __syncthreads();
    }

    // Second half of matrix filling.
    for (int i_pattern = 1; i_pattern < numRows; ++i_pattern)
    {
        // Advance one diag.
        auto tmp = _prevScores;
        _prevScores = _thisScores;
        _thisScores = _prevPrevScores;
        _prevPrevScores = tmp;

        const int threadInRowIdx = numCols-1 - tid + i_pattern;

        if (tid >= i_pattern)
        {
            const char textByte = textBytes[threadInRowIdx - 1];
            const int scoreMatrixIdx = ((int) textByte) * alphabetSize + ((int) patternByte);

            const int fromLeftScore = _prevScores[tid] - gapPenalty;
            const int fromTopScore = _prevScores[tid - 1] - gapPenalty;
            const int fromDiagScore = _prevPrevScores[tid - 1] + _scoreMatrix[scoreMatrixIdx];

            const int maxWithGap = max(fromLeftScore, fromTopScore);
            _thisScores[tid] = max(maxWithGap, fromDiagScore);

            const auto dirWithGap = (fromTopScore > fromLeftScore) ? DIR::TOP : DIR::LEFT;
            M[tid*numCols + threadInRowIdx] = (fromDiagScore > maxWithGap) ? DIR::DIAG : dirWithGap;
        }

        if (tid == (diag_size-1))
            yield(columnState, workerId, threadInRowIdx);

        __syncthreads();
    }

    if (tid == (numRows - 1))
        finalScore[0] = _thisScores[tid];
}


void SequenceAlignment::alignSequenceGlobalGPU(const SequenceAlignment::Request &request,
                                               SequenceAlignment::Response *response)
{
    const unsigned int numCols = request.textNumBytes + 1;
    const unsigned int numRows = request.patternNumBytes + 1;
    char *M;

    /** Allocate host memory */
    try
    {
        M = new char[numRows * numCols];
        response->alignedTextBytes = new char[2 * request.textNumBytes];
        response->alignedPatternBytes = new char[2 * request.textNumBytes];
    }
    catch(const std::bad_alloc& e)
    {
        std::cerr << SequenceAlignment::MEM_ERROR;
        return;
    }
    /** End Allocate host memory */

    int *d_finalScore, *d_scoreMatrix;
    char *d_textBytes, *d_patternBytes, *d_M, *d_columnState;

    auto freeMemory = [&]()
    {
        hipFree(d_finalScore);
        hipFree(d_scoreMatrix);
        hipFree(d_M);
        hipFree(d_textBytes);
        hipFree(d_patternBytes);
        hipFree(d_columnState);
        delete [] M;
    };

    /** Allocate and transfer memory to device */
    if (hipMalloc(&d_finalScore, sizeof(int)) != hipSuccess ||
        hipMalloc(&d_scoreMatrix, sizeof(int) * request.alphabetSize * request.alphabetSize) != hipSuccess ||
        hipMalloc(&d_M, numRows * numCols) != hipSuccess ||
        hipMalloc(&d_textBytes, request.textNumBytes) != hipSuccess ||
        hipMalloc(&d_patternBytes, request.patternNumBytes) != hipSuccess ||
        hipMalloc(&d_columnState, numCols) != hipSuccess)
    {
        std::cout << MEM_ERROR << std::endl;
        freeMemory();
        return;
    }
    if (hipMemcpy(d_textBytes, request.textBytes, request.textNumBytes, hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_patternBytes, request.patternBytes, request.patternNumBytes, hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_scoreMatrix, request.scoreMatrix, sizeof(int) * (request.alphabetSize * request.alphabetSize), hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cout << MEM_ERROR << std::endl;
        freeMemory();
        return;
    }
    /** End Allocate and transfer memory */

    const unsigned int sharedMemSize = 3 * numRows * sizeof(int) +
                                       request.alphabetSize * request.alphabetSize * sizeof(int);

    int startRow = 0;
    for (int i=0; i < (numRows/MAX_THREADS_PER_BLOCK + 1); ++i)
    {
        const int numThreads = std::min(MAX_THREADS_PER_BLOCK, numRows - startRow);
        const int endRow = startRow + numThreads;

        const int workerId = (i%2 == 0) ? 0 : 1;
        cuda_fillMatrixNW<<<1, numThreads, sharedMemSize>>>(d_textBytes, d_patternBytes,
                                                            d_scoreMatrix, request.alphabetSize,
                                                            request.gapPenalty, startRow, endRow,
                                                            numCols, workerId, d_columnState,
                                                            d_M, d_finalScore);

        startRow = std::min(startRow + MAX_THREADS_PER_BLOCK, numRows-1);
    }

    // std::cout << "Num rows: " << numRows << "\n";
    // std::cout << "Num col: " << numCols << "\n";
    // std::cout << "Num bytes in shared: " << sharedMemSize << "\n";

    if (hipMemcpy(&(response->score), d_finalScore, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(M, d_M, numRows*numCols, hipMemcpyDeviceToHost) != hipSuccess)
    {
        std::cout << "Could not copy back to host memory" << std::endl;
        return;
    }

    traceBack(M, numRows, numCols, request, response);

    freeMemory();

}
