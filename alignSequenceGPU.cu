#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "SequenceAlignment.hpp"



__global__ void alignSequenceGlobalCUDA(const char *textBytes, const uint64_t textNumBytes,
                                        const char *patternBytes, const uint64_t patternNumBytes,
                                        const char *alphabet, const int alphabetSize,
                                        const short *scoreMatrix, const short gapPenalty,
                                        const int numRows, const int numCols,
                                        int *thisScores, int *prevScores)
{
    extern __shared__ int _shared[];
    int *_thisScores = _shared;
    int *_prevScores = _shared + numCols;

    const int tid = threadIdx.x;

    // Each thread copies one text letter.
    const char textByte = (tid > 0) ? textBytes[tid - 1] : alphabetSize;
    // Init first row.
    _thisScores[tid] = tid * -gapPenalty;

    // Dynamic programming loop.
    for (int i_pattern = 1; i_pattern < numRows; ++i_pattern)
    {
        // Advance one row.
         int *tmp = _thisScores;
        _thisScores = _prevScores;
        _prevScores = tmp;

        if (tid == 0)
        {
            _thisScores[tid] = _prevScores[tid] - gapPenalty;
            continue;
        }

        const char patternByte = patternBytes[i_pattern - 1];
        const int scoreMatrixIdx = ((int) textByte) * alphabetSize + ((int) patternByte);

        // We are accessing the previous row - wait for all columns to finish.
        __syncthreads();
        const int fromTopScore = _prevScores[tid] - gapPenalty;
        const int fromDiagScore = _prevScores[tid - 1] + scoreMatrix[scoreMatrixIdx];

        const int maxFromPrev = max(fromDiagScore, fromTopScore);

        for (int i_text = 1; i_text < numCols; ++i_text)
        {
            // We are accessing the previous column within a row.
            if (tid == i_text)
            {
                const int fromLeftScore = _thisScores[tid - 1] - gapPenalty;
                _thisScores[tid] = max(maxFromPrev, fromLeftScore);
            }
            __syncthreads();
        }
    }

    if (tid == (numCols - 1))
        thisScores[tid] = _thisScores[tid];
}


void SequenceAlignment::alignSequenceGlobalGPU(const SequenceAlignment::Request &request,
                                               SequenceAlignment::Response *response)
{
    const unsigned int numCols = request.textNumBytes + 1;
    const unsigned int numRows = request.patternNumBytes + 1;

    int *d_thisScores, *d_prevScores;
    short *d_scoreMatrix;
    char *d_textBytes, *d_patternBytes, *d_alphabet;

    if (hipMalloc(&d_thisScores, sizeof(int) * numCols) != hipSuccess ||
        hipMalloc(&d_prevScores, sizeof(int) * numCols) != hipSuccess ||
        hipMalloc(&d_scoreMatrix, sizeof(short) * request.alphabetSize * request.alphabetSize) != hipSuccess ||
        hipMalloc(&d_textBytes, request.textNumBytes) != hipSuccess ||
        hipMalloc(&d_patternBytes, request.patternNumBytes) != hipSuccess ||
        hipMalloc(&d_alphabet, request.alphabetSize) != hipSuccess)
    {
        std::cout << MEM_ERROR << std::endl;
        hipFree(d_thisScores);
        hipFree(d_prevScores);
        hipFree(d_scoreMatrix);
        hipFree(d_textBytes);
        hipFree(d_patternBytes);
        return;
    }

    if (hipMemcpy(d_textBytes, request.textBytes, request.textNumBytes, hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_patternBytes, request.patternBytes, request.patternNumBytes, hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_scoreMatrix, request.scoreMatrix, sizeof(short) * (request.alphabetSize * request.alphabetSize), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_alphabet, request.alphabet, request.alphabetSize, hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cout << MEM_ERROR << std::endl;
        hipFree(d_thisScores);
        hipFree(d_prevScores);
        hipFree(d_scoreMatrix);
        hipFree(d_textBytes);
        hipFree(d_patternBytes);
        return;
    }

    // this and prev row scores
    const unsigned int sharedMemSize = 2 * sizeof(int) * numCols;
    std::cout << "Num bytes in col: " << sharedMemSize << "\n";
    std::cout << "Num cols: " << numCols << "\n";

    alignSequenceGlobalCUDA<<<1, numCols, sharedMemSize>>>(d_textBytes, request.textNumBytes,
                                                           d_patternBytes, request.patternNumBytes,
                                                           d_alphabet, request.alphabetSize,
                                                           d_scoreMatrix, request.gapPenalty,
                                                           numRows, numCols,
                                                           d_thisScores, d_prevScores);

    if (hipMemcpy(&(response->score), (d_thisScores + numCols - 1), sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
        std::cout << "Could not copy back to host memory" << std::endl;

    hipFree(d_thisScores);
    hipFree(d_prevScores);
    hipFree(d_scoreMatrix);
    hipFree(d_textBytes);
    hipFree(d_patternBytes);

    std::cout << "# Score: \t" << response->score << "\n";

}
