#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "SequenceAlignment.hpp"

constexpr unsigned int MAX_THREADS_PER_BLOCK = 1024;

using SequenceAlignment::DIR;

struct columnState { int score; int kernelId; };

__device__ __forceinline__ void set_done(columnState* volatile colState, const int col,
                                         const int score, const int kernelId)
{
    colState[col] = {score, kernelId+1};
}

__device__ __forceinline__ void busy_wait(columnState* volatile colState, const int col,
                                          const int kernelId)
{
    while (colState[col].kernelId != kernelId) continue;
}

__device__ __forceinline__ void ping_pong_buffers(int *&first, int *&second, int *&third)
{
    auto tmpSecond = second;
    second = first;
    first = third;
    third = tmpSecond;
}

__device__ __forceinline__ int choose_direction(const int leftScore,
                                                const int topScore,
                                                const int diagScore,
                                                const int gapPenalty,
                                                const int letterScore,
                                                char *directionStore)
{
    const int fromLeftScore = leftScore - gapPenalty;
    const int fromTopScore = topScore - gapPenalty;
    const int fromDiagScore = diagScore + letterScore;

    const int maxWithGap = max(fromLeftScore, fromTopScore);
    const int maxOverall = max(maxWithGap, fromDiagScore);

    const auto dirWithGap = (fromTopScore > fromLeftScore) ? DIR::TOP : DIR::LEFT;
    directionStore[0] = (fromDiagScore > maxWithGap) ? DIR::DIAG : dirWithGap;

    return maxOverall;
}

__global__ void cuda_fillMatrixNW(const char *textBytes, const char *patternBytes,
                                  const int *scoreMatrix, const int alphabetSize, const int gapPenalty,
                                  const int startRow, const int endRow, const int numCols,
                                  const int kernelId, columnState* colState, char *M)
{
    const int numRows = blockDim.x;

    extern __shared__ int _shared[];
    int *_scoreMatrix = _shared;
    int *_thisScores = _shared + alphabetSize*alphabetSize;
    int *_prevScores = _thisScores + numRows;
    int *_prevPrevScores = _prevScores + numRows;

    const int tid = threadIdx.x;

    // Transfer score matrix to shared memory.
    for (int offset=0; offset < alphabetSize*alphabetSize; offset += numRows)
    {
        if ((offset + tid) < alphabetSize*alphabetSize)
            _scoreMatrix[offset + tid] = scoreMatrix[offset + tid];
    }

    const char patternByte = patternBytes[max(0, tid - 1 + startRow)];

    __syncthreads();

    // First half of matrix filling
    int diag_size = 0;
    for (int i_text = 0; i_text < numCols; ++i_text)
    {
        ping_pong_buffers(_thisScores, _prevScores, _prevPrevScores);

        diag_size = min(diag_size+1, numRows);
        const int threadInRowIdx = i_text - tid;

        if (startRow > 0)
            busy_wait(colState, i_text, kernelId);

        if (tid == 0 && startRow == 0)
        {
            // First row.
            _thisScores[tid] = -(i_text * gapPenalty);
            M[i_text] = DIR::LEFT;
        }
        else if (tid == (diag_size-1) && i_text < numRows)
        {
            // First column.
            _thisScores[tid] = -((tid + startRow) * gapPenalty);
            M[tid * numCols] = DIR::TOP;
        }
        else if (tid < diag_size)
        {
            const char textByte = textBytes[threadInRowIdx - 1];
            const int scoreMatrixIdx = ((int) textByte) * alphabetSize + ((int) patternByte);

            _thisScores[tid] = choose_direction(_prevScores[tid],
                                                _prevScores[tid - 1],
                                                _prevPrevScores[tid - 1],
                                                gapPenalty, _scoreMatrix[scoreMatrixIdx],
                                                (M + tid*numCols + threadInRowIdx));
        }
        __syncthreads();

        if (tid == endRow && tid < diag_size)
            set_done(colState, threadInRowIdx, _thisScores[tid], kernelId);
    }

    // Second half of matrix filling.
    for (int i_pattern = 1; i_pattern < numRows; ++i_pattern)
    {
        ping_pong_buffers(_thisScores, _prevScores, _prevPrevScores);

        const int threadInRowIdx = numCols-1 - tid + i_pattern;

        if (tid >= i_pattern)
        {
            const char textByte = textBytes[threadInRowIdx - 1];
            const int scoreMatrixIdx = ((int) textByte) * alphabetSize + ((int) patternByte);

            _thisScores[tid] = choose_direction(_prevScores[tid],
                                                _prevScores[tid - 1],
                                                _prevPrevScores[tid - 1],
                                                gapPenalty, _scoreMatrix[scoreMatrixIdx],
                                                (M + tid*numCols + threadInRowIdx));

            if (tid == endRow)
                set_done(colState, threadInRowIdx, _thisScores[tid], kernelId);
        }

        __syncthreads();
    }

}


void SequenceAlignment::alignSequenceGlobalGPU(const SequenceAlignment::Request &request,
                                               SequenceAlignment::Response *response)
{
    const unsigned int numCols = request.textNumBytes + 1;
    const unsigned int numRows = request.patternNumBytes + 1;
    char *M;

    /** Allocate host memory */
    try
    {
        M = new char[numRows * numCols];
        response->alignedTextBytes = new char[2 * request.textNumBytes];
        response->alignedPatternBytes = new char[2 * request.textNumBytes];
    }
    catch(const std::bad_alloc& e)
    {
        std::cerr << SequenceAlignment::MEM_ERROR;
        return;
    }
    /** End Allocate host memory */

    char *d_textBytes, *d_patternBytes, *d_M;
    int *d_scoreMatrix;
    columnState *d_columnState;

    auto freeMemory = [&]()
    {
        hipFree(d_textBytes);
        hipFree(d_patternBytes);
        hipFree(d_M);
        hipFree(d_scoreMatrix);
        hipFree(d_columnState);
        delete [] M;
    };

    /** Allocate and transfer memory to device */
    if (hipMalloc(&d_scoreMatrix, sizeof(int) * request.alphabetSize * request.alphabetSize) != hipSuccess ||
        hipMalloc(&d_M, numRows * numCols) != hipSuccess ||
        hipMalloc(&d_textBytes, request.textNumBytes) != hipSuccess ||
        hipMalloc(&d_patternBytes, request.patternNumBytes) != hipSuccess ||
        hipMalloc(&d_columnState, numCols * sizeof(columnState)) != hipSuccess)
    {
        std::cout << MEM_ERROR << std::endl;
        freeMemory();
        return;
    }
    if (hipMemcpy(d_textBytes, request.textBytes, request.textNumBytes, hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_patternBytes, request.patternBytes, request.patternNumBytes, hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_scoreMatrix, request.scoreMatrix, sizeof(int) * (request.alphabetSize * request.alphabetSize), hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cout << MEM_ERROR << std::endl;
        freeMemory();
        return;
    }
    /** End Allocate and transfer memory */

    const unsigned int sharedMemSize = 3 * numRows * sizeof(int) +
                                       request.alphabetSize * request.alphabetSize * sizeof(int);

    int startRow = 0;
    for (int i_kernel=0; i_kernel < (numRows/MAX_THREADS_PER_BLOCK + 1); ++i_kernel)
    {
        const int numThreads = std::min(MAX_THREADS_PER_BLOCK, numRows - startRow);
        const int endRow = startRow + numThreads - 1;

        cuda_fillMatrixNW<<<1, numThreads, sharedMemSize>>>(d_textBytes, d_patternBytes,
                                                            d_scoreMatrix, request.alphabetSize,
                                                            request.gapPenalty, startRow, endRow,
                                                            numCols, i_kernel, d_columnState, d_M);

        startRow = endRow + 1;
    }

    // std::cout << "Num rows: " << numRows << "\n";
    // std::cout << "Num col: " << numCols << "\n";
    // std::cout << "Num bytes in shared: " << sharedMemSize << "\n";

    hipDeviceSynchronize();
    if (hipMemcpy(&(response->score), &(d_columnState[numCols - 1].score), sizeof(int), hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(M, d_M, numRows*numCols, hipMemcpyDeviceToHost) != hipSuccess)
    {
        std::cout << "Could not copy back to host memory" << std::endl;
        freeMemory();
        return;
    }

    traceBack(M, numRows, numCols, request, response);

    freeMemory();

}
