#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "SequenceAlignment.hpp"

constexpr uint64_t MAX_THREADS_PER_BLOCK = 1024;

using SequenceAlignment::DIR;

struct columnState { int score; int kernelId; };

__device__ __forceinline__ void set_done(columnState* volatile colState, const unsigned int col,
                                         const int score, const int kernelId)
{
    colState[col] = {score, kernelId+1};
}

__device__ __forceinline__ void busy_wait(columnState* volatile colState, const unsigned int col,
                                          const int kernelId)
{
    volatile int currKernelId = colState[col].kernelId;
    while (currKernelId != kernelId)
        currKernelId = colState[col].kernelId;
}

__device__ __forceinline__ void ping_pong_buffers(int *&first, int *&second, int *&third)
{
    auto tmpSecond = second;
    second = first;
    first = third;
    third = tmpSecond;
}

__device__ __forceinline__ int choose_direction(const int leftScore,
                                                const int topScore,
                                                const int diagScore,
                                                const int gapPenalty,
                                                const int letterScore,
                                                char *directionStore)
{
    const int fromLeftScore = leftScore - gapPenalty;
    const int fromTopScore = topScore - gapPenalty;
    const int fromDiagScore = diagScore + letterScore;

    const int maxWithGap = max(fromLeftScore, fromTopScore);
    const int maxOverall = max(maxWithGap, fromDiagScore);

    const auto dirWithGap = (fromTopScore > fromLeftScore) ? DIR::TOP : DIR::LEFT;
    *directionStore = (fromDiagScore > maxWithGap) ? DIR::DIAG : dirWithGap;

    return maxOverall;
}

__global__ void cuda_fillMatrixNW(const char* __restrict__ textBytes,
                                  const char* __restrict__ patternBytes,
                                  const int* __restrict__ scoreMatrix,
                                  const int alphabetSize, const int gapPenalty,
                                  const int startRow, const int endRow, const int numCols,
                                  const int kernelId,
                                  columnState* __restrict__ colState,
                                  char* __restrict__ M)
{
    const int numRows = blockDim.x;
    const int tid = threadIdx.x;

    extern __shared__ int _shared[];
    int *_scoreMatrix = _shared;
    int *_thisScores = _shared + alphabetSize*alphabetSize;
    int *_prevScores = _thisScores + numRows;
    int *_prevPrevScores = _prevScores + numRows;

    // Transfer score matrix to shared memory.
    for (int offset=0; offset < alphabetSize * alphabetSize; offset += numRows)
    {
        if (offset + tid < alphabetSize * alphabetSize)
            _scoreMatrix[offset + tid] = scoreMatrix[offset + tid];
    }

    // Each thread gets one row (one pattern letter).
    const char patternByte = patternBytes[max(0, (tid + startRow) - 1)];
    M[tid * numCols] = DIR::TOP;

    // First half of matrix filling
    int diag_size = 0;
    int fromLeft, fromDiag;
    // fromDiag_{iteration_i} = fromTop_{iteration_i - 1}
    int fromTop = (tid + startRow-1) * gapPenalty;
    for (int i_text = 1; i_text < numCols; ++i_text)
    {
        ping_pong_buffers(_thisScores, _prevScores, _prevPrevScores);

        diag_size = min(diag_size + 1, numRows);
        const int idxInRow = i_text - tid;

        if (tid == 0)
            busy_wait(colState, i_text, kernelId);

        // Wait for thread 0 and for _prevScores from previous iteration to be updated.
        __syncthreads();

        if (tid < diag_size)
        {
            fromDiag = fromTop;
            fromLeft = (idxInRow == 1) ? (tid + startRow) * gapPenalty : _prevScores[tid];
            fromTop = (tid == 0) ? colState[i_text].score : _prevScores[max(0, tid - 1)];

            const char textByte = textBytes[idxInRow - 1];
            const int scoreMatrixIdx = ((int) textByte) * alphabetSize + ((int) patternByte);

            _thisScores[tid] = choose_direction(fromLeft, fromTop, fromDiag,
                                                gapPenalty, _scoreMatrix[scoreMatrixIdx],
                                                (M + tid*numCols + idxInRow));

            if ((tid + startRow) == endRow)
                set_done(colState, idxInRow, _thisScores[tid], kernelId);
        }
    }

    // Second half of matrix filling.
    for (int i_pattern = 1; i_pattern < numRows; ++i_pattern)
    {
        ping_pong_buffers(_thisScores, _prevScores, _prevPrevScores);

        const int idxInRow = (numCols - 1) + (i_pattern - tid);

        __syncthreads();

        if (tid >= i_pattern)
        {
            const char textByte = textBytes[idxInRow - 1];
            const int scoreMatrixIdx = ((int) textByte) * alphabetSize + ((int) patternByte);

            _thisScores[tid] = choose_direction(_prevScores[tid],
                                                _prevScores[tid - 1],
                                                _prevPrevScores[tid - 1],
                                                gapPenalty, _scoreMatrix[scoreMatrixIdx],
                                                (M + tid*numCols + idxInRow));

            if ((tid + startRow) == endRow)
                set_done(colState, idxInRow, _thisScores[tid], kernelId);
        }
    }

    __syncthreads();
}


uint64_t initMemory(const SequenceAlignment::Request &request, SequenceAlignment::Response *response,
                    char *&d_M0, char *&d_M1, char *&d_textBytes,
                    char *&d_patternBytes, int *&d_scoreMatrix, columnState *&d_columnState,
                    char *&h_M0, char *&h_M1, int *&h_score, char *&os_M, hipStream_t &cuStream)
{
    const uint64_t numCols = request.textNumBytes + 1;
    const uint64_t numRows = request.patternNumBytes + 1;

    try
    {
        os_M = new char[numRows * numCols];
        response->alignedTextBytes = new char[2 * request.textNumBytes];
        response->alignedPatternBytes = new char[2 * request.textNumBytes];
    }
    catch(const std::bad_alloc& e)
    {
        return 0;
    }

    auto numBytesGlobalGPU = [&] (int numThreads)
    {
        return sizeof(int) * request.alphabetSize * request.alphabetSize +  // scoreMatrix
               2 * numThreads * numCols +                                   // M0, M1
               request.textNumBytes + request.patternNumBytes +             // sequences
               sizeof(columnState) * numCols;                               // columState
    };

    // Select a number of threads per block such that we fit into global memory.
    uint64_t numThreads = MAX_THREADS_PER_BLOCK;
    uint64_t freeGlobalMem = 0;
    hipMemGetInfo((size_t*) &freeGlobalMem, 0);
    while (freeGlobalMem < numBytesGlobalGPU(numThreads))
    {
        numThreads -= 32;
        if (numThreads < 32)
            return 0;
    }

    if (hipMalloc(&d_scoreMatrix, sizeof(int) * request.alphabetSize * request.alphabetSize) != hipSuccess ||
        hipMalloc(&d_M0, numThreads * numCols) != hipSuccess ||
        hipMalloc(&d_M1, numThreads * numCols) != hipSuccess ||
        hipMalloc(&d_textBytes, request.textNumBytes) != hipSuccess ||
        hipMalloc(&d_patternBytes, request.patternNumBytes) != hipSuccess ||
        hipMalloc(&d_columnState, numCols * sizeof(columnState)) != hipSuccess ||
        hipHostMalloc(&h_M0, numThreads * numCols) != hipSuccess ||
        hipHostMalloc(&h_M1, numThreads * numCols) != hipSuccess ||
        hipHostMalloc(&h_score, sizeof(int)) != hipSuccess)
    {
        return 0;
    }

    // Initialize the very first row scores and directions.
    std::vector<columnState> initState(numCols);
    for (int i=0; i<numCols; ++i)
    {
        initState[i].score = i * request.gapPenalty;
        initState[i].kernelId = 0;
    }
    std::fill_n(os_M, numCols, DIR::LEFT);

    if (hipMemcpyAsync(d_textBytes, request.textBytes, request.textNumBytes, hipMemcpyHostToDevice, cuStream) != hipSuccess ||
        hipMemcpyAsync(d_patternBytes, request.patternBytes, request.patternNumBytes, hipMemcpyHostToDevice, cuStream) != hipSuccess ||
        hipMemcpyAsync(d_scoreMatrix, request.scoreMatrix, sizeof(int) * (request.alphabetSize * request.alphabetSize), hipMemcpyHostToDevice, cuStream) != hipSuccess ||
        hipMemcpyAsync(d_columnState, &(initState[0]), sizeof(columnState) * numCols, hipMemcpyHostToDevice, cuStream) != hipSuccess)
    {
        return 0;
    }

    hipStreamSynchronize(cuStream);

    return numThreads;
}

int SequenceAlignment::alignSequenceGlobalGPU(const SequenceAlignment::Request &request,
                                               SequenceAlignment::Response *response)
{
    const uint64_t numCols = request.textNumBytes + 1;
    const uint64_t numRows = request.patternNumBytes + 1;

    // Use 2 streams to..
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    // Used to identify current stream for CUDA operations. Start with stream0.
    hipStream_t currStream = stream0;

    /** Memory allocation and transfer. */
    // OS managed memory (can be swapped to disk).
    char *os_M;
    // CUDA managed host memory, pinned to physical mem address and not swappable.
    char *h_M0, *h_M1;
    int *h_score;
    // Device memory.
    char *d_textBytes, *d_patternBytes, *d_M0, *d_M1;
    int *d_scoreMatrix;
    columnState *d_columnState;

    auto cleanUp = [&]()
    {
        hipStreamDestroy(stream0);
        hipStreamDestroy(stream1);

        if (d_textBytes) hipFree(d_textBytes);
        if (d_patternBytes) hipFree(d_patternBytes);
        if (d_M0) hipFree(d_M0);
        if (d_M1) hipFree(d_M1);
        if (d_scoreMatrix) hipFree(d_scoreMatrix);
        if (d_columnState) hipFree(d_columnState);

        if (h_M0) hipHostFree(h_M0);
        if (h_M1) hipHostFree(h_M1);
        if (h_score) hipHostFree(h_score);

        if (os_M) delete [] os_M;
    };

    const uint64_t NUM_THREADS_PER_BLOCK = initMemory(request, response, d_M0, d_M1, d_textBytes,
                                                      d_patternBytes, d_scoreMatrix, d_columnState,
                                                      h_M0, h_M1, h_score, os_M, currStream);
    if (NUM_THREADS_PER_BLOCK == 0)
    {
        std::cout << MEM_ERROR;
        cleanUp();
        return -1;
    }
    /** End Allocate and transfer memory */

    const unsigned int sharedMemSize = 3 * std::min(NUM_THREADS_PER_BLOCK, numRows) * sizeof(int) +
                                       request.alphabetSize * request.alphabetSize * sizeof(int);

    #ifdef BENCHMARK
        auto begin = std::chrono::steady_clock::now();
    #endif

    // First row is already inititalized, start from second.
    int startRow = 1;
    auto curr_os_M = os_M + numCols;
    for (int i_kernel=0; i_kernel < (numRows/NUM_THREADS_PER_BLOCK + 1); ++i_kernel)
    {
        const int numThreads = std::min(NUM_THREADS_PER_BLOCK, numRows - startRow);
        const int endRow = startRow + numThreads - 1;

        currStream = (i_kernel % 2 == 0) ? stream0 : stream1;
        auto curr_d_M = (i_kernel % 2 == 0) ? d_M0 : d_M1;
        auto curr_h_M = (i_kernel % 2 == 0) ? h_M0 : h_M1;

        cuda_fillMatrixNW<<<1, numThreads, sharedMemSize, currStream>>>
            (d_textBytes, d_patternBytes, d_scoreMatrix, request.alphabetSize, request.gapPenalty,
             startRow, endRow, numCols, i_kernel, d_columnState, curr_d_M);

        // Get the filled out part of M matrix in this iteration.
        // From device memory -> CUDA managed pinned memory -> OS managed swappable memory.
        if (hipMemcpyAsync(curr_h_M, curr_d_M, numThreads*numCols, hipMemcpyDeviceToHost, currStream) != hipSuccess)
        {
            std::cout << "error: could not copy from device memory\n";
            hipDeviceSynchronize();
            cleanUp();
            return -1;
        }

        hipStreamSynchronize(currStream);
        std::copy_n(curr_h_M, numThreads*numCols, curr_os_M);
        curr_os_M += numThreads*numCols;

        startRow = endRow + 1;
    }

    if (hipMemcpyAsync(h_score, &(d_columnState[numCols - 1].score), sizeof(int), hipMemcpyDeviceToHost, currStream) != hipSuccess)
    {
        std::cout << "error: could not copy from device memory\n";
        cleanUp();
        return -1;
    }

    hipStreamSynchronize(currStream);
    response->score = *h_score;

    #ifdef BENCHMARK
        // If benchmraking, return the time taken instead of error code.
        auto end = std::chrono::steady_clock::now();
        cleanUp();
        return std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    #endif

    traceBack(os_M, numRows, numCols, request, response);

    cleanUp();

    return 0;
}
