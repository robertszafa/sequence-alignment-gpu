#include "hip/hip_runtime.h"
#include "SequenceAlignment.hpp"

#include <hip/hip_runtime.h>


__global__ void alignSequenceGlobalCUDA(const char *textBytes, const uint64_t textNumBytes,
                                        const char *patternBytes, const uint64_t patternNumBytes,
                                        const char *alphabet, const int alphabetSize,
                                        const short *scoreMatrix, const short gap_penalty,
                                        int *thisScores, int *prevScores)
{
}


void SequenceAlignment::alignSequenceGlobalGPU(const SequenceAlignment::Request &request,
                                               SequenceAlignment::Response *response)
{
    const unsigned int numCols = request.textNumBytes + 1;
    const unsigned int numRows = request.patternNumBytes + 1;

    int *d_thisScores, *d_prevScores;
    short *d_scoreMatrix;
    char *d_textBytes, *d_patternBytes, *d_alphabet;

    if (hipMalloc(&d_thisScores, sizeof(int) * numCols) != hipSuccess ||
        hipMalloc(&d_prevScores, sizeof(int) * numCols) != hipSuccess ||
        hipMalloc(&d_scoreMatrix, sizeof(short) * request.alphabetSize) != hipSuccess ||
        hipMalloc(&d_textBytes, request.textNumBytes) != hipSuccess ||
        hipMalloc(&d_patternBytes, request.patternNumBytes) != hipSuccess ||
        hipMalloc(&d_alphabet, request.alphabetSize) != hipSuccess)
    {
        std::cout << MEM_ERROR << std::endl;
        hipFree(d_thisScores);
        hipFree(d_prevScores);
        hipFree(d_scoreMatrix);
        hipFree(d_textBytes);
        hipFree(d_patternBytes);
        return;
    }

    if (hipMemcpy(d_textBytes, &request.textBytes, request.textNumBytes, hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_patternBytes, &request.patternBytes, request.patternNumBytes, hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_scoreMatrix, &request.scoreMatrix, (request.alphabetSize * request.alphabetSize), hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_alphabet, &request.alphabet, request.alphabetSize, hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cout << MEM_ERROR << std::endl;
        hipFree(d_thisScores);
        hipFree(d_prevScores);
        hipFree(d_scoreMatrix);
        hipFree(d_textBytes);
        hipFree(d_patternBytes);
        return;
    }

    alignSequenceGlobalCUDA<<<1, numCols>>>(d_textBytes, request.textNumBytes,
                                            d_patternBytes, request.patternNumBytes,
                                            d_alphabet, request.alphabetSize,
                                            d_scoreMatrix, request.gapPenalty,
                                            d_thisScores, d_prevScores);

    if (hipMemcpy(&(d_thisScores + numCols - 1), response->score, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
        std::cout << "Could not copy back to host memory" << std::endl;

    hipFree(d_thisScores);
    hipFree(d_prevScores);
    hipFree(d_scoreMatrix);
    hipFree(d_textBytes);
    hipFree(d_patternBytes);

    std::cout << "# Score: \t" << response.score << "\n";

}
