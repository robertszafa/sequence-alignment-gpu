#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "SequenceAlignment.hpp"

constexpr uint64_t MAX_THREADS_PER_BLOCK = 1024;

using SequenceAlignment::DIR;

struct columnState { int score; int kernelId; };

__device__ __forceinline__ void set_done(columnState* volatile colState, const unsigned int col,
                                         const int score, const int kernelId)
{
    colState[col] = {score, kernelId+1};
}

__device__ __forceinline__ void busy_wait(columnState* volatile colState, const unsigned int col,
                                          const int kernelId)
{
    if (threadIdx.x == 0)
    {
        volatile int currKernelId = colState[col].kernelId;
        while (currKernelId != kernelId)
            currKernelId = colState[col].kernelId;
    }
}

__device__ __forceinline__ void ping_pong_buffers(int *&first, int *&second, int *&third)
{
    auto tmpSecond = second;
    second = first;
    first = third;
    third = tmpSecond;
}

__device__ __forceinline__ int choose_direction(const int leftScore,
                                                const int topScore,
                                                const int diagScore,
                                                const int gapPenalty,
                                                const int letterScore,
                                                char *directionStore)
{
    const int fromLeftScore = leftScore - gapPenalty;
    const int fromTopScore = topScore - gapPenalty;
    const int fromDiagScore = diagScore + letterScore;

    const int maxWithGap = max(fromLeftScore, fromTopScore);
    const int maxOverall = max(maxWithGap, fromDiagScore);

    const auto dirWithGap = (fromTopScore > fromLeftScore) ? DIR::TOP : DIR::LEFT;
    *directionStore = (fromDiagScore > maxWithGap) ? DIR::DIAG : dirWithGap;

    return maxOverall;
}

__global__ void cuda_fillMatrixNW(const char *textBytes, const char *patternBytes,
                                  const int *scoreMatrix, const int alphabetSize, const int gapPenalty,
                                  const int startRow, const int endRow, const int numCols,
                                  const int kernelId, columnState* colState, char *M)
{
    const int numRows = blockDim.x;

    extern __shared__ int _shared[];
    int *_scoreMatrix = _shared;
    int *_thisScores = _shared + alphabetSize*alphabetSize;
    int *_prevScores = _thisScores + numRows;
    int *_prevPrevScores = _prevScores + numRows;

    const int tid = threadIdx.x;

    // Transfer score matrix to shared memory.
    for (int offset=0; offset < alphabetSize*alphabetSize; offset += numRows)
    {
        if ((offset + tid) < alphabetSize*alphabetSize)
            _scoreMatrix[offset + tid] = scoreMatrix[offset + tid];
    }

    const char patternByte = patternBytes[max(0, (tid+startRow) - 1)];

    // First half of matrix filling
    int diag_size = 0;
    for (int i_text = 0; i_text < numCols; ++i_text)
    {
        ping_pong_buffers(_thisScores, _prevScores, _prevPrevScores);

        diag_size = min(diag_size+1, numRows);
        const int threadInRowIdx = i_text - tid;

        if (startRow > 0)
            busy_wait(colState, i_text, kernelId);

        __syncthreads();

        if (tid == 0 && startRow == 0) // First row.
        {
            _thisScores[tid] = -(i_text * gapPenalty);
            M[i_text] = DIR::LEFT;
        }
        else if (tid == (diag_size-1) && i_text < numRows) // First column.
        {
            _thisScores[tid] = -((tid + startRow) * gapPenalty);
            M[(tid+startRow) * numCols] = DIR::TOP;
        }
        else if (tid == 0) // Not first row of M, but first row in this kernel.
        {
            const char textByte = textBytes[threadInRowIdx - 1];
            const int scoreMatrixIdx = ((int) textByte) * alphabetSize + ((int) patternByte);

            _thisScores[tid] = choose_direction(_prevScores[tid],
                                                colState[i_text].score,
                                                colState[i_text - 1].score,
                                                gapPenalty, _scoreMatrix[scoreMatrixIdx],
                                                (M + (tid+startRow)*numCols + threadInRowIdx));
        }
        else if (tid < diag_size)
        {
            const char textByte = textBytes[threadInRowIdx - 1];
            const int scoreMatrixIdx = ((int) textByte) * alphabetSize + ((int) patternByte);

            _thisScores[tid] = choose_direction(_prevScores[tid],
                                                _prevScores[tid - 1],
                                                _prevPrevScores[tid - 1],
                                                gapPenalty, _scoreMatrix[scoreMatrixIdx],
                                                (M + (tid+startRow)*numCols + threadInRowIdx));
        }

        if ((tid + startRow) == endRow && tid < diag_size)
            set_done(colState, threadInRowIdx, _thisScores[tid], kernelId);
    }

    // Second half of matrix filling.
    for (int i_pattern = 1; i_pattern < numRows; ++i_pattern)
    {
        ping_pong_buffers(_thisScores, _prevScores, _prevPrevScores);

        const int threadInRowIdx = numCols-1 - tid + i_pattern;

        __syncthreads();

        if (tid >= i_pattern)
        {
            const char textByte = textBytes[threadInRowIdx - 1];
            const int scoreMatrixIdx = ((int) textByte) * alphabetSize + ((int) patternByte);

            _thisScores[tid] = choose_direction(_prevScores[tid],
                                                _prevScores[tid - 1],
                                                _prevPrevScores[tid - 1],
                                                gapPenalty, _scoreMatrix[scoreMatrixIdx],
                                                (M + (tid+startRow)*numCols + threadInRowIdx));

            if ((tid + startRow) == endRow)
                set_done(colState, threadInRowIdx, _thisScores[tid], kernelId);
        }
    }
}


int SequenceAlignment::alignSequenceGlobalGPU(const SequenceAlignment::Request &request,
                                               SequenceAlignment::Response *response)
{
    const uint64_t numCols = request.textNumBytes + 1;
    const uint64_t numRows = request.patternNumBytes + 1;

    // Memory for aligned sequences.
    try
    {
        response->alignedTextBytes = new char[2 * request.textNumBytes];
        response->alignedPatternBytes = new char[2 * request.textNumBytes];
    }
    catch(const std::bad_alloc& e)
    {
        std::cerr << SequenceAlignment::MEM_ERROR;
        return -1;
    }

    /** Allocate and transfer memory to GPU. */
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    char *h_M;
    int *h_score;

    char *d_textBytes, *d_patternBytes, *d_M;
    int *d_scoreMatrix;
    columnState *d_columnState;

    auto cleanUp = [&]()
    {
        hipStreamDestroy(stream0);
        hipStreamDestroy(stream1);

        if (d_textBytes) hipFree(d_textBytes);
        if (d_patternBytes) hipFree(d_patternBytes);
        if (d_M) hipFree(d_M);
        if (d_scoreMatrix) hipFree(d_scoreMatrix);
        if (d_columnState) hipFree(d_columnState);

        if (h_M) hipHostFree(h_M);
        if (h_score) hipHostFree(h_score);
    };

    if (hipMalloc(&d_scoreMatrix, sizeof(int) * request.alphabetSize * request.alphabetSize) != hipSuccess ||
        hipMalloc(&d_M, numRows * numCols) != hipSuccess ||
        hipMalloc(&d_textBytes, request.textNumBytes) != hipSuccess ||
        hipMalloc(&d_patternBytes, request.patternNumBytes) != hipSuccess ||
        hipMalloc(&d_columnState, numCols * sizeof(columnState)) != hipSuccess ||
        hipHostMalloc(&h_M, numRows * numCols) != hipSuccess ||
        hipHostMalloc(&h_score, sizeof(int)) != hipSuccess)
    {
        std::cout << MEM_ERROR << std::endl;
        cleanUp();
        return -1;
    }
    if (hipMemcpyAsync(d_textBytes, request.textBytes, request.textNumBytes, hipMemcpyHostToDevice, stream0) != hipSuccess ||
        hipMemcpyAsync(d_patternBytes, request.patternBytes, request.patternNumBytes, hipMemcpyHostToDevice, stream0) != hipSuccess ||
        hipMemcpyAsync(d_scoreMatrix, request.scoreMatrix, sizeof(int) * (request.alphabetSize * request.alphabetSize), hipMemcpyHostToDevice, stream0) != hipSuccess)
    {
        std::cout << MEM_ERROR << std::endl;
        cleanUp();
        return -1;
    }

    hipMemsetAsync(d_columnState, 0, sizeof(columnState) * numCols, stream0);
    /** End Allocate and transfer memory */


    #ifdef BENCHMARK
        auto begin = std::chrono::steady_clock::now();
    #endif

    const unsigned int sharedMemSize = 3 * std::min(MAX_THREADS_PER_BLOCK, numRows) * sizeof(int) +
                                       request.alphabetSize * request.alphabetSize * sizeof(int);

    int startRow = 0;
    hipStream_t currStream;
    for (int i_kernel=0; i_kernel < (numRows/MAX_THREADS_PER_BLOCK + 1); ++i_kernel)
    {
        const int numThreads = std::min(MAX_THREADS_PER_BLOCK, numRows - startRow);
        const int endRow = startRow + numThreads - 1;

        currStream = (i_kernel % 2 == 0) ? stream0 : stream1;

        cuda_fillMatrixNW<<<1, numThreads, sharedMemSize, currStream>>>
            (d_textBytes, d_patternBytes, d_scoreMatrix, request.alphabetSize, request.gapPenalty,
             startRow, endRow, numCols, i_kernel, d_columnState, d_M);

        startRow = endRow + 1;
    }

    if (hipMemcpyAsync(h_score, &(d_columnState[numCols - 1].score), sizeof(int), hipMemcpyDeviceToHost, currStream) != hipSuccess ||
        hipMemcpyAsync(h_M, d_M, numRows*numCols, hipMemcpyDeviceToHost, currStream) != hipSuccess)
    {
        std::cout << "Could not copy back to host memory" << std::endl;
        cleanUp();
        return -1;
    }

    hipStreamSynchronize(currStream);
    response->score = *h_score;

    #ifdef BENCHMARK
        auto end = std::chrono::steady_clock::now();
        cleanUp();
        return std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    #endif

    traceBack(h_M, numRows, numCols, request, response);

    cleanUp();

    return 0;
}
