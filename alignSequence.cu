#include "SequenceAlignment.hpp"

#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>

void parseArguments(int argc, char *argv[])
{
    // We need at least the text and pattern file.
    if (argc < 3) std::cerr << "Usage: " << SequenceAlignment::USAGE << std::endl;

    for (int i = 1; i < argc; ++i)
    {
        if (argv[i][0] == '-' && strlen(argv[i]) > 1)
        {
            if (SequenceAlignment::argumentMap.count(argv[i][1]) > 0)
            {
                auto setArg = SequenceAlignment::argumentMap.at(argv[i][1]);
                SequenceAlignment::deviceType = (setArg == SequenceAlignment::programArgs::CPU) ||
                                                (setArg == SequenceAlignment::programArgs::GPU)
                                                ? setArg
                                                : SequenceAlignment::deviceType;
                SequenceAlignment::sequenceType = (setArg == SequenceAlignment::programArgs::DNA) ||
                                                  (setArg == SequenceAlignment::programArgs::PROTEIN)
                                                  ? setArg
                                                  : SequenceAlignment::sequenceType;
            }
            else
            {
                std::cerr << "Ignoring \"" << argv[i] << "\"" << std::endl;
            }
        }
        else
        {
            std::ifstream f(argv[i]);
            if (f.good())
            {
                // Use string's range constructor to copy over entire file to memory.
                std::string fileString((std::istreambuf_iterator<char>(f)),
                                        std::istreambuf_iterator<char>());

                if (SequenceAlignment::textNumBytes == 0)
                {
                    SequenceAlignment::textBytes = fileString.c_str();
                    SequenceAlignment::textNumBytes = fileString.length();
                }
                else
                {
                    SequenceAlignment::patternBytes = fileString.c_str();
                    SequenceAlignment::patternNumBytes = fileString.length();
                }

            }
            else
            {
                std::cout << argv[i] << " file does not exist" << std::endl;
            }

            f.close();
        }
    }
}

int main(int argc, char *argv[])
{
    parseArguments(argc, argv);

    std::cout << SequenceAlignment::textNumBytes << " " << SequenceAlignment::patternNumBytes << "\n";
}
